#include "hip/hip_runtime.h"
/**
* Copyright 2016 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <stdio.h>


// Reference outputs (calculated on an M40 GPU)
// > ./RNN 20 2 512 64 0
// Forward: 1299 GFLOPs
// Backward: 2171 GFLOPs, (1564 GFLOPs), (3549 GFLOPs)
// i checksum 1.315793E+06     h checksum 1.315212E+05
// di checksum 6.676003E+01    dh checksum 6.425067E+01
// dw checksum 1.453750E+09
//
// > ./RNN 20 2 512 64 1
// Forward: 1296 GFLOPs
// Backward: 2235 GFLOPs, (1567 GFLOPs), (3896 GFLOPs)
// i checksum 6.319591E+05     h checksum 6.319605E+04
// di checksum 4.501830E+00    dh checksum 4.489546E+00
// dw checksum 5.012598E+07
//
// > ./RNN 20 2 512 64 2
// Forward: 2635 GFLOPs
// Backward: 2757 GFLOPs, (2001 GFLOPs), (4433 GFLOPs)
// i checksum 5.749536E+05     c checksum 4.365091E+05     h checksum 5.774818E+04
// di checksum 3.842206E+02    dc checksum 9.323785E+03    dh checksum 1.182566E+01
// dw checksum 4.313461E+08
//
// > ./RNN 20 2 512 64 3
// Forward: 2428 GFLOPs
// Backward: 2645 GFLOPs, (1915 GFLOPs), (4270 GFLOPs)
// i checksum 6.358978E+05     h checksum 6.281680E+04
// di checksum 6.296622E+00    dh checksum 2.289960E+05
// dw checksum 5.397419E+07



// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cudnnErrCheck(stat) { cudnnErrCheck_((stat), __FILE__, __LINE__); }
void cudnnErrCheck_(hipdnnStatus_t stat, const char *file, int line) {
   if (stat != HIPDNN_STATUS_SUCCESS) {
      fprintf(stderr, "cuDNN Error: %s %s %d\n", hipdnnGetErrorString(stat), file, line);
   }
}

__global__ void initGPUData_ker(float *data, int numElements, float value) {
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < numElements) {
      data[tid] = value;
   }
}

void initGPUData(float *data, int numElements, float value) {
   dim3 gridDim;
   dim3 blockDim;
   
   blockDim.x = 1024;
   gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;
   
   initGPUData_ker <<< gridDim, blockDim >>> (data, numElements, value);
}

  
int main(int argc, char* argv[]) {

   int seqLength;
   int numLayers;
   int hiddenSize;
   int inputSize;
   int miniBatch;
   float dropout;
   bool bidirectional;
   int mode;
   int persistent;

   FILE *fp;
   fp=fopen("result.txt","w");

   if (argc == 6) {
      seqLength = atoi(argv[1]);
      numLayers = atoi(argv[2]);
      hiddenSize = atoi(argv[3]);
      inputSize = hiddenSize;
      miniBatch = atoi(argv[4]);
      dropout = 0;
      bidirectional = 0;
      mode = atoi(argv[5]);
      persistent = 0;
   }
   else {
      printf("Usage:\n");
      printf("./RNN <seqLength> <numLayers> <hiddenSize> <miniBatch> <mode>\n");
      printf("Modes: 0 = RNN_RELU, 1 = RNN_TANH, 2 = LSTM, 3 = GRU\n");
      return 1;
   }

   // -------------------------   
   // Create cudnn context
   // -------------------------  
   hipdnnHandle_t cudnnHandle;   
   cudnnErrCheck(hipdnnCreate(&cudnnHandle));

   
   // -------------------------   
   // Set up inputs and outputs
   // -------------------------
   void *x;
   void *hx = NULL;
   void *cx = NULL;
   
   void *dx;
   void *dhx = NULL;
   void *dcx = NULL;
  
   void *y;
   void *hy = NULL;
   void *cy = NULL;
   
   void *dy;
   void *dhy = NULL;
   void *dcy = NULL;
   
   // Memory allocation. hx, cx, dhx, dcx, hy, cy, dhy and dcy can be NULL.
   cudaErrCheck(hipMalloc((void**)&x, seqLength * inputSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&hx, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&cx, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
   
   cudaErrCheck(hipMalloc((void**)&dx, seqLength * inputSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&dhx, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&dcx, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
   
   cudaErrCheck(hipMalloc((void**)&y, seqLength * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&hy, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&cy, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
   
   cudaErrCheck(hipMalloc((void**)&dy, seqLength * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&dhy, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&dcy, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1) * sizeof(float)));
      
   // Set up tensor descriptors. x/y/dx/dy are arrays, one per time step.
   hipdnnTensorDescriptor_t *xDesc, *yDesc, *dxDesc, *dyDesc;
   hipdnnTensorDescriptor_t hxDesc, cxDesc;
   hipdnnTensorDescriptor_t hyDesc, cyDesc;
   hipdnnTensorDescriptor_t dhxDesc, dcxDesc;
   hipdnnTensorDescriptor_t dhyDesc, dcyDesc;
   
   xDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
   yDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
   dxDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
   dyDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
   
   int dimA[3];
   int strideA[3];

   // In this example dimA[1] is constant across the whole sequence
   // This isn't required, all that is required is that it does not increase.
   for (int i = 0; i < seqLength; i++) {
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&xDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&yDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&dxDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&dyDesc[i]));
   
      dimA[0] = miniBatch;
      dimA[1] = inputSize;
      dimA[2] = 1;
     
      strideA[0] = dimA[2] * dimA[1];
      strideA[1] = dimA[2];
      strideA[2] = 1;

      cudnnErrCheck(hipdnnSetTensorNdDescriptor(xDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(dxDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
      
      dimA[0] = miniBatch;
      dimA[1] = bidirectional ? hiddenSize * 2 : hiddenSize;
      dimA[2] = 1;

      strideA[0] = dimA[2] * dimA[1];
      strideA[1] = dimA[2];
      strideA[2] = 1;
      
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(yDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(dyDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   }
   
   
   dimA[0] = numLayers * (bidirectional ? 2 : 1);
   dimA[1] = miniBatch;
   dimA[2] = hiddenSize;
   
   strideA[0] = dimA[2] * dimA[1];
   strideA[1] = dimA[2];
   strideA[2] = 1;
   
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&hxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&cxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&hyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&cyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcyDesc));
   
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(hxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(cxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(hyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(cyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
  
  
   // -------------------------
   // Set up the dropout descriptor (needed for the RNN descriptor)
   // -------------------------
   unsigned long long seed = 1337ull; // Pick a seed.
   
   hipdnnDropoutDescriptor_t dropoutDesc;
   cudnnErrCheck(hipdnnCreateDropoutDescriptor(&dropoutDesc));
   
   // How much memory does dropout need for states?
   // These states are used to generate random numbers internally
   // and should not be freed until the RNN descriptor is no longer used
   size_t stateSize;
   void *states;
   cudnnErrCheck(hipdnnDropoutGetStatesSize(cudnnHandle, &stateSize));
   
   cudaErrCheck(hipMalloc(&states, stateSize));
   
   cudnnErrCheck(hipdnnSetDropoutDescriptor(dropoutDesc, 
                             cudnnHandle,
                             dropout, 
                             states, 
                             stateSize, 
                             seed));
                             
   // -------------------------   
   // Set up the RNN descriptor
   // -------------------------
   hipdnnRNNDescriptor_t rnnDesc;
   hipdnnRNNMode_t RNNMode;
   hipdnnRNNAlgo_t RNNAlgo;
   
   cudnnErrCheck(hipdnnCreateRNNDescriptor(&rnnDesc));
   
   if      (mode == 0) RNNMode = HIPDNN_RNN_RELU;
   else if (mode == 1) RNNMode = HIPDNN_RNN_TANH;
   else if (mode == 2) RNNMode = HIPDNN_LSTM;
   else if (mode == 3) RNNMode = HIPDNN_GRU;
   
   // Persistent RNNs are only supported on Pascal+ GPUs.
   if      (persistent == 0) RNNAlgo = HIPDNN_RNN_ALGO_STANDARD;
   else if (persistent == 1) RNNAlgo = HIPDNN_RNN_ALGO_PERSIST_STATIC;
   else if (persistent == 2) RNNAlgo = HIPDNN_RNN_ALGO_PERSIST_DYNAMIC;
      
   cudnnErrCheck(hipdnnSetRNNDescriptor_v6(cudnnHandle,
                                       rnnDesc,
                                       hiddenSize, 
                                       numLayers, 
                                       dropoutDesc,
                                       HIPDNN_LINEAR_INPUT, // We can also skip the input matrix transformation
                                       bidirectional ? HIPDNN_BIDIRECTIONAL : HIPDNN_UNIDIRECTIONAL, 
                                       RNNMode, 
                                       RNNAlgo, // Can be changed to use persistent RNNs on Pascal+ GPUs.
                                       HIPDNN_DATA_FLOAT));
   
   
   // -------------------------
   // Set up parameters
   // -------------------------
   // This needs to be done after the rnn descriptor is set as otherwise
   // we don't know how many parameters we have to allocate
   void *w;   
   void *dw;   

   hipdnnFilterDescriptor_t wDesc, dwDesc;
   
   cudnnErrCheck(hipdnnCreateFilterDescriptor(&wDesc));
   cudnnErrCheck(hipdnnCreateFilterDescriptor(&dwDesc));
   
   size_t weightsSize;
   cudnnErrCheck(hipdnnGetRNNParamsSize(cudnnHandle, rnnDesc, xDesc[0], &weightsSize, HIPDNN_DATA_FLOAT));
   
   int dimW[3];   
   dimW[0] =  weightsSize / sizeof(float);
   dimW[1] = 1;
   dimW[2] = 1;
      
   cudnnErrCheck(hipdnnSetFilterNdDescriptor(wDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, dimW));   
   cudnnErrCheck(hipdnnSetFilterNdDescriptor(dwDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, dimW));   
   
   cudaErrCheck(hipMalloc((void**)&w,  weightsSize));
   cudaErrCheck(hipMalloc((void**)&dw, weightsSize));
   
   
   // -------------------------
   // Set up work space and reserved memory
   // -------------------------   
   void *workspace;
   void *reserveSpace;   
   
   size_t workSize;
   size_t reserveSize;

   // Need for every pass
   cudnnErrCheck(hipdnnGetRNNWorkspaceSize(cudnnHandle, rnnDesc, seqLength, xDesc, &workSize));
   // Only needed in training, shouldn't be touched between passes.
   cudnnErrCheck(hipdnnGetRNNTrainingReserveSize(cudnnHandle, rnnDesc, seqLength, xDesc, &reserveSize));
    
   cudaErrCheck(hipMalloc((void**)&workspace, workSize));
   cudaErrCheck(hipMalloc((void**)&reserveSpace, reserveSize));
   
   // *********************************************************************************************************
   // Initialise weights and inputs
   // *********************************************************************************************************
   // We initialise to something simple.
   // Matrices are initialised to 1 / matrixSize, biases to 1, data is 1.
   initGPUData((float*)x, seqLength * inputSize * miniBatch, 1.f);
   if (hx != NULL) initGPUData((float*)hx, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1), 1.f);
   if (cx != NULL) initGPUData((float*)cx, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1), 1.f);
   
   initGPUData((float*)dy, seqLength * hiddenSize * miniBatch * (bidirectional ? 2 : 1), 1.f);
   if (dhy != NULL) initGPUData((float*)dhy, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1), 1.f);
   if (dcy != NULL) initGPUData((float*)dcy, numLayers * hiddenSize * miniBatch * (bidirectional ? 2 : 1), 1.f);
      
   
   // Weights
   int numLinearLayers = 0;
   if (RNNMode == HIPDNN_RNN_RELU || RNNMode == HIPDNN_RNN_TANH) {
      numLinearLayers = 2;
   }
   else if (RNNMode == HIPDNN_LSTM) {
      numLinearLayers = 8;
   }
   else if (RNNMode == HIPDNN_GRU) {
      numLinearLayers = 6;
   }
   
   for (int layer = 0; layer < numLayers * (bidirectional ? 2 : 1); layer++) {
      for (int linLayerID = 0; linLayerID < numLinearLayers; linLayerID++) {
         hipdnnFilterDescriptor_t linLayerMatDesc;
         cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerMatDesc));
         float *linLayerMat;
         
         cudnnErrCheck(hipdnnGetRNNLinLayerMatrixParams( cudnnHandle,
                                                        rnnDesc,  
                                                        layer,
                                                        xDesc[0], 
                                                        wDesc, 
                                                        w,
                                                        linLayerID,  
                                                        linLayerMatDesc, 
                                                        (void**)&linLayerMat));
         
         hipdnnDataType_t dataType;
         hipdnnTensorFormat_t format;
         int nbDims;
         int filterDimA[3];
         cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerMatDesc,
                                                  3,
                                                  &dataType,
                                                  &format,
                                                  &nbDims,
                                                  filterDimA));
                                                  
         initGPUData(linLayerMat, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f / (float)(filterDimA[0] * filterDimA[1] * filterDimA[2]));                                                 

         cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerMatDesc));         
         
         hipdnnFilterDescriptor_t linLayerBiasDesc;
         cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerBiasDesc));
         float *linLayerBias;
         
         cudnnErrCheck(hipdnnGetRNNLinLayerBiasParams( cudnnHandle,
                                                        rnnDesc,  
                                                        layer,
                                                        xDesc[0], 
                                                        wDesc, 
                                                        w,
                                                        linLayerID,  
                                                        linLayerBiasDesc, 
                                                        (void**)&linLayerBias));
         
         cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerBiasDesc,
                                                  3,
                                                  &dataType,
                                                  &format,
                                                  &nbDims,
                                                  filterDimA));
                                                  
         initGPUData(linLayerBias, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f);
                                                  
         cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerBiasDesc));
      }
   }
   
   // *********************************************************************************************************
   // Dynamic persistent RNN plan (if using this algo)
   // *********************************************************************************************************
   hipdnnPersistentRNNPlan_t rnnPlan;
   if (RNNAlgo == HIPDNN_RNN_ALGO_PERSIST_DYNAMIC) {
      // Note: This step is expensive. Once completed the plan can be reused so long as the descriptor
      //       minibatch or datatype don't change.
      cudnnErrCheck(hipdnnCreatePersistentRNNPlan(rnnDesc, miniBatch, HIPDNN_DATA_FLOAT, &rnnPlan));
      // Tell calls using this descriptor which plan to use.
      cudnnErrCheck(hipdnnSetPersistentRNNPlan(rnnDesc, rnnPlan));
   }
   
   // *********************************************************************************************************
   // At this point all of the setup is done. We now need to pass through the RNN.
   // *********************************************************************************************************
   cudaErrCheck(hipDeviceSynchronize());
   
   hipEvent_t start, stop;
   float timeForward, timeBackward1, timeBackward2;
   cudaErrCheck(hipEventCreate(&start));
   cudaErrCheck(hipEventCreate(&stop));
   
   cudaErrCheck(hipEventRecord(start));   

   // If we're not training we use this instead
   // cudnnErrCheck(hipdnnRNNForwardInference(cudnnHandle, 
                                         // rnnDesc, 
                                         // seqLength,                                          
                                         // xDesc, 
                                         // x, 
                                         // hxDesc,
                                         // hx, 
                                         // cxDesc, 
                                         // cx, 
                                         // wDesc, 
                                         // w, 
                                         // yDesc,  
                                         // y, 
                                         // hyDesc, 
                                         // hy, 
                                         // cyDesc, 
                                         // cy, 
                                         // workspace, 
                                         // workSize));

   cudnnErrCheck(hipdnnRNNForwardTraining(cudnnHandle, 
                                         rnnDesc, 
                                         seqLength,                                          
                                         xDesc, 
                                         x, 
                                         hxDesc,
                                         hx, 
                                         cxDesc, 
                                         cx, 
                                         wDesc, 
                                         w, 
                                         yDesc,  
                                         y, 
                                         hyDesc, 
                                         hy, 
                                         cyDesc, 
                                         cy, 
                                         workspace, 
                                         workSize,
                                         reserveSpace, 
                                         reserveSize));
                
   cudaErrCheck(hipEventRecord(stop));   
   cudaErrCheck(hipEventSynchronize(stop));
   cudaErrCheck(hipEventElapsedTime(&timeForward, start, stop));
   
   cudaErrCheck(hipEventRecord(start));
   
   cudnnErrCheck(hipdnnRNNBackwardData(cudnnHandle, 
                               rnnDesc, 
                               seqLength,                                
                               yDesc, 
                               y,
                               dyDesc, 
                               dy, 
                               dhyDesc, 
                               dhy, 
                               dcyDesc, 
                               dcy, 
                               wDesc, 
                               w, 
                               hxDesc, 
                               hx,
                               cxDesc, 
                               cx,
                               dxDesc, 
                               dx, 
                               dhxDesc,
                               dhx,
                               dcxDesc,
                               dcx,
                               workspace,
                               workSize,
                               reserveSpace, 
                               reserveSize ));
   
   cudaErrCheck(hipEventRecord(stop));   
   cudaErrCheck(hipEventSynchronize(stop));
   cudaErrCheck(hipEventElapsedTime(&timeBackward1, start, stop));
   
   cudaErrCheck(hipEventRecord(start));
   
   // hipdnnRNNBackwardWeights adds to the data in dw.
   cudaErrCheck(hipMemset(dw, 0, weightsSize));
   
   cudnnErrCheck(hipdnnRNNBackwardWeights( cudnnHandle, 
                                    rnnDesc, 
                                    seqLength, 
                                    xDesc, 
                                    x, 
                                    hxDesc, 
                                    hx,                                                   
                                    yDesc, 
                                    y,
                                    workspace, 
                                    workSize, 
                                    dwDesc, 
                                    dw,
                                    reserveSpace, 
                                    reserveSize ));
                     
                     

   cudaErrCheck(hipEventRecord(stop));   

   cudaErrCheck(hipEventSynchronize(stop));
   cudaErrCheck(hipEventElapsedTime(&timeBackward2, start, stop));

   
   int numMats = 0;
   
   if (RNNMode == HIPDNN_RNN_RELU || RNNMode == HIPDNN_RNN_TANH) {
      numMats = 2;
   }
   else if (RNNMode == HIPDNN_LSTM) {
      numMats = 8;
   }
   else if (RNNMode == HIPDNN_GRU) {
      numMats = 6;
   }
   
   // Calculate FLOPS
   printf("Forward: %3.0f GFLOPS\n", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeForward));
   printf("Backward: %3.0f GFLOPS, ", numMats * 4ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * (timeBackward1 + timeBackward2)));
   printf("(%3.0f GFLOPS), ", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward1));
   printf("(%3.0f GFLOPS)\n", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward2));

   // Calculate FLOPS
   fprintf(fp,"Forward: %3.0f GFLOPS\n", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeForward));
   fprintf(fp,"Backward: %3.0f GFLOPS, ", numMats * 4ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * (timeBackward1 + timeBackward2)));
   fprintf(fp,"(%3.0f GFLOPS), ", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward1));
   fprintf(fp,"(%3.0f GFLOPS)\n", numMats * 2ull * (bidirectional ? 2 : 1) * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward2));

   // Make double-sure everything is finished before we copy for result checking.
   hipDeviceSynchronize();
   
   // *********************************************************************************************************
   // Print checksums.
   // *********************************************************************************************************
   if (true) {
      float* testOutputi;
      float* testOutputh;
      float* testOutputc;
      
      int biDirScale = (bidirectional ? 2 : 1);
      
      testOutputi = (float*)malloc(hiddenSize * seqLength * miniBatch * biDirScale * sizeof(float));
      testOutputh = (float*)malloc(hiddenSize * miniBatch * numLayers * biDirScale * sizeof(float));
      testOutputc = (float*)malloc(hiddenSize * miniBatch * numLayers * biDirScale * sizeof(float));
 
      cudaErrCheck(hipMemcpy(testOutputi, y, hiddenSize * seqLength * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      if (hy != NULL) cudaErrCheck(hipMemcpy(testOutputh, hy, numLayers * hiddenSize * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      if (cy != NULL && RNNMode == HIPDNN_LSTM) cudaErrCheck(hipMemcpy(testOutputc, cy, numLayers * hiddenSize * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      
      double checksumi = 0.f;
      double checksumh = 0.f;
      double checksumc = 0.f;
      
      for (int m = 0; m < miniBatch; m++) {
         double localSumi = 0;
         double localSumh = 0;
         double localSumc = 0;
         
         for (int j = 0; j < seqLength; j++) {
            for (int i = 0; i < hiddenSize * biDirScale; i++) {   
               localSumi += testOutputi[j * miniBatch * hiddenSize * biDirScale + m * hiddenSize * biDirScale + i];
            }
         }
         for (int j = 0; j < numLayers * biDirScale; j++) {
            for (int i = 0; i < hiddenSize; i++) {         
               if (hy != NULL) localSumh += testOutputh[j * hiddenSize * miniBatch + m * hiddenSize + i];
               if (cy != NULL) if (RNNMode == HIPDNN_LSTM) localSumc += testOutputc[j * hiddenSize * miniBatch + m * hiddenSize + i];
            }
         }
                  
         checksumi += localSumi;
         checksumh += localSumh;
         checksumc += localSumc;
      }
      
      printf("i checksum %E     ", checksumi);
      fprintf(fp,"i checksum %E     ", checksumi);
      if (RNNMode == HIPDNN_LSTM) { printf("c checksum %E     ", checksumc); fprintf(fp,"c checksum %E     ", checksumc); }
      printf("h checksum %E\n", checksumh);
      fprintf(fp,"h checksum %E\n", checksumh);
      
      free(testOutputi);
      free(testOutputc);
      free(testOutputh);
   }   
   
   if (true) {
      float* testOutputdi;
      float* testOutputdh;
      float* testOutputdc;

      int biDirScale = (bidirectional ? 2 : 1);
      
      testOutputdi = (float*)malloc(inputSize * seqLength * miniBatch * sizeof(float));
      testOutputdh = (float*)malloc(hiddenSize * miniBatch * numLayers * biDirScale * sizeof(float));
      testOutputdc = (float*)malloc(hiddenSize * miniBatch * numLayers * biDirScale * sizeof(float));
      cudaErrCheck(hipMemcpy(testOutputdi, dx, seqLength * miniBatch * inputSize * sizeof(float), hipMemcpyDeviceToHost));
      if (dhx != NULL) cudaErrCheck(hipMemcpy(testOutputdh, dhx, numLayers * hiddenSize * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      if (dcx != NULL) if (RNNMode == HIPDNN_LSTM) cudaErrCheck(hipMemcpy(testOutputdc, dcx, numLayers * hiddenSize * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      
      float checksumdi = 0.f;
      float checksumdh = 0.f;
      float checksumdc = 0.f;
      
      for (int m = 0; m < miniBatch; m++) {
         double localSumdi = 0;
         double localSumdh = 0;
         double localSumdc = 0;

         for (int j = 0; j < seqLength; j++) {
            for (int i = 0; i < inputSize; i++) {
               localSumdi += testOutputdi[j * miniBatch * inputSize + m * inputSize + i];
            }
         }

         for (int j = 0; j < numLayers * biDirScale; j++) {
            for (int i = 0; i < hiddenSize; i++) {         
               localSumdh += testOutputdh[j * hiddenSize * miniBatch + m * hiddenSize + i];
               if (RNNMode == HIPDNN_LSTM) localSumdc += testOutputdc[j * hiddenSize * miniBatch + m * hiddenSize + i];
            }
         }         

         checksumdi += localSumdi;
         checksumdh += localSumdh;
         checksumdc += localSumdc;
         
      }
      
      printf("di checksum %E    ", checksumdi);
      fprintf(fp,"di checksum %E    ", checksumdi);
      if (RNNMode == HIPDNN_LSTM) { printf("dc checksum %E    ", checksumdc); fprintf(fp,"dc checksum %E    ", checksumdc); }
      printf("dh checksum %E\n", checksumdh);
      fprintf(fp,"dh checksum %E\n", checksumdh);
      
      free(testOutputdi);
      free(testOutputdh);
      free(testOutputdc);
   }

   if (true) {
      float* testOutputdw;
      testOutputdw = (float*)malloc(weightsSize);
 
      cudaErrCheck(hipMemcpy(testOutputdw, dw, weightsSize, hipMemcpyDeviceToHost));
      
      double checksumdw = 0.;
            
      for (int i = 0; i < weightsSize / sizeof(float); i++) {
         checksumdw += testOutputdw[i];
      }
      
      printf("dw checksum %E\n", checksumdw);
      fprintf(fp,"dw checksum %E\n", checksumdw);
      
      free(testOutputdw);
   }

   if (RNNAlgo == HIPDNN_RNN_ALGO_PERSIST_DYNAMIC) {
      hipdnnDestroyPersistentRNNPlan(rnnPlan);
   }  
  
   hipFree(x);
   hipFree(hx);
   hipFree(cx);
   hipFree(y);
   hipFree(hy);
   hipFree(cy);
   hipFree(dx);
   hipFree(dhx);
   hipFree(dcx);
   hipFree(dy);
   hipFree(dhy);
   hipFree(dcy);
   hipFree(workspace);
   hipFree(reserveSpace);
   hipFree(w);
   hipFree(dw);
   
   hipdnnDestroy(cudnnHandle);
   fclose(fp);
   return 0;
}



